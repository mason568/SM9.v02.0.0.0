#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
// Example code from the Udacity tutorial on CUDA                        //
// Link to the video here: https://www.youtube.com/watch?v=GiGE3QjwknQ   //
// TO COMPILE: $ nvcc -o square square.cu                                //
///////////////////////////////////////////////////////////////////////////

//
// NOTES:
//  * Device: is the term of the GPU
//  * Host: is the term of the CPU
//  * kernels are the only things run in parallel on the GPU
//  * everything in the main is run on CPU
//  * memory transfers between Host (CPU) and Device (GPU) should be minimal
//  * kernels all run at the same time
//  * threads can know their Id's with threadIdx.x, blocks are similar
//

#include <stdio.h>
#include "gputimer.h"
#include "square063.cuh"

// kernel to be run on the TX2
__global__ void square(float *d_out, float *d_in){
  int idx = threadIdx.x; // this is how you get the thread index
  float f = d_in[idx];
  d_out[idx] = f*f;
}



// kernel to be run on the GTX 3090

// main is here. this is the CPU code. 

void squaretest(){

  // the size of the array, which is really the thread count per block
  const int ARRAY_SIZE = 1024; // max thread count per block on the TX2
  // total bytes in the array
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  // generate the input array on the host for the device my dude
  float h_in[ARRAY_SIZE];
  for (int i = 0; i < ARRAY_SIZE; i++){
    h_in[i] = float(i);
  }
  float h_out[ARRAY_SIZE]; // make the other array too my guy
  
  // make the pointers that point to the mem locations on the GPU m'lady
  float *d_in;
  float *d_out;

  //allocate the memory on the GPU my man
  hipMalloc((void **) &d_in, ARRAY_BYTES);
  hipMalloc((void **) &d_out, ARRAY_BYTES);

  // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

  // launch the kernel
  square<<<1,ARRAY_SIZE>>>(d_out,d_in);

  // copy the result back to the CPU mem
  hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

  // print the result
  for (int i = 0; i < ARRAY_SIZE; i++){
    printf("%f", h_out[i]);
    printf(((i % 4) != 3) ? "\t" : "\n");
  }
  // free the device yo
  hipFree(d_in);
  hipFree(d_out);
}

