#include "hip/hip_runtime.h"
#include "Bigintcuda.cuh"



//构造大数对象并初始化为零 + 其对应的kernal函数

__global__ void cbiginiinit_thread(CBigInt *d_out){
    int idx = threadIdx.x; // this is how you get the thread index
	d_out[idx].m_nLength=1;
	for(int i=0;i<BI_MAXLEN;i++)
		d_out[idx].m_ulValue[i]=0;
}

void CBigIntInit_para(CBigInt *A,int parasize)
{
    CBigInt *h_A, *dev_A;


    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_A,parasize*(sizeof(CBigInt)),hipHostMallocDefault));

    
	CHECK(hipMalloc((void **)&dev_A,parasize*(sizeof(CBigInt))));
    // copy from host memory to pinned memory

    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    //hipMemcpy(dev_A, h_A, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    cbiginiinit_thread<<<1,parasize>>>(dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_A, dev_A, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    // print the result
    
    for (int i = 0; i < parasize; i++){
        printf("para %d = %s\n",i,Put(h_A[i],HEX));
    }

    Mov_Big_Big(A,h_A[0]);

    // free the device yo
    hipFree(h_A);
    hipFree(dev_A);
}


/****************************************************************************************
大数比较
调用方式：Cmp_para(N,A,num)
返回值：若N<A返回-1；若N=A返回0；若N>A返回1

****************************************************************************************/
__global__ void cmp_thread(CBigInt *dev_N,CBigInt *dev_A, int *d_cmp_res){
    int idx = threadIdx.x; // this is how you get the thread index



	int i;
	if(dev_N[idx].m_nLength>dev_A[idx].m_nLength){
		d_cmp_res[idx] =1;
        return;
    }
	if(dev_N[idx].m_nLength<dev_A[idx].m_nLength){
		d_cmp_res[idx] =-1;
        return;
    }
	for(i=dev_N[idx].m_nLength-1;i>=0;i--)
	{
		if(dev_N[idx].m_ulValue[i]>dev_A[idx].m_ulValue[i]){
            d_cmp_res[idx] =1;
            return;
        }
		if(dev_N[idx].m_ulValue[i]<dev_A[idx].m_ulValue[i]){
            d_cmp_res[idx] =-1;
            return;
        }
	}
    d_cmp_res[idx] = 0;
	return;
}

int Cmp_para(CBigInt N,CBigInt A, int parasize)
{

    CBigInt *h_N, *h_A, *dev_N,*dev_A;
    int *h_cmp_ret, *d_cmp_ret;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    for(int i=0;i<parasize;i++){
        Mov_Big_Big(&h_N[i],N);
        Mov_Big_Big(&h_A[i],A);
    }
    CHECK(hipHostAlloc((void**) &h_cmp_ret,parasize*(sizeof(int)),hipHostMallocDefault));
    CHECK(hipMalloc((void **)&dev_N,parasize*(sizeof(CBigInt))));
    CHECK(hipMalloc((void **)&dev_A,parasize*(sizeof(CBigInt))));
    CHECK(hipMalloc((void **)&d_cmp_ret,parasize*(sizeof(int))));
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    cmp_thread<<<1,parasize>>>(dev_N,dev_A,d_cmp_ret);

    // copy the result back to the CPU mem
    hipMemcpy(h_cmp_ret, d_cmp_ret, parasize*(sizeof(int)), hipMemcpyDeviceToHost);
    for(int i=0;i<parasize;i++){
        printf("para %d cmp_ret = %d\n",i, h_cmp_ret[i]);
    }
    return h_cmp_ret[0];
}

/****************************************************************************************
大数赋值
调用方式：assignn_Big_to_Big_para(&N,A,32)  
返回值：多个N,被赋值为同一个A
****************************************************************************************/
__global__ void mov_big_big_thread(CBigInt *dev_N, CBigInt *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
    dev_N[idx].m_nLength=dev_A[idx].m_nLength;

	for(int i=0;i<BI_MAXLEN;i++)
	{
		dev_N[idx].m_ulValue[i]=dev_A[idx].m_ulValue[i];
	}
}
__global__ void assignn_Big_to_Big_thread(CBigInt *dev_N, CBigInt *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
    dev_N[idx].m_nLength=dev_A->m_nLength;

	for(int i=0;i<BI_MAXLEN;i++)
	{
		dev_N[idx].m_ulValue[i]=dev_A->m_ulValue[i];
	}
}

void assignn_Big_to_Big_para(CBigInt *N, CBigInt A, int parasize)
{
    CBigInt *h_N, *dev_N;
    CBigInt *h_A,*dev_A;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    //CHECK(hipHostAlloc((void**) &h_A,sizeof(int),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));
    Mov_Big_Big(h_A,A);
    CHECK(hipMalloc((void **)&dev_N,parasize*(sizeof(CBigInt))));
    //CHECK(hipMalloc((void **)&dev_A,parasize*(sizeof(CBigInt))));
	CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    // copy from host memory to pinned memory
    
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(CBigInt), hipMemcpyHostToDevice);

    // launch the kernel
    assignn_Big_to_Big_thread<<<1,parasize>>>(dev_N,dev_A);
    
    // copy the result back to the CPU mem
    hipMemcpy(h_N, dev_N, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);
   
    // print the result
    
    for (int i = 0; i < parasize; i++){
        printf("thread %d move A=%s to N=%s\n",i,Put(A,HEX),Put(h_N[i],HEX));
    }

    Mov_Big_Big(N,h_N[0]);
   
    // free the device yo
    hipFree(h_N);
    hipFree(dev_N);
    hipFree(h_A);
    hipFree(dev_A);

}


void Mov_Big_Big_para(CBigInt *N, CBigInt A, int parasize)
{
    CBigInt *h_N, *dev_N, *h_AA, *dev_AA;
    //CBigInt h_A,dev_A;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_AA,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    for(int i=0;i<parasize;i++){
        Mov_Big_Big(&h_AA[i],A);
    }
    //CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));

    CHECK(hipMalloc((void **)&dev_N,parasize*(sizeof(CBigInt))));
    CHECK(hipMalloc((void **)&dev_AA,parasize*(sizeof(CBigInt))));
	//CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    // copy from host memory to pinned memory
    
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_AA, h_AA, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    mov_big_big_thread<<<1,parasize>>>(dev_N,dev_AA);
   
    // copy the result back to the CPU mem
    hipMemcpy(h_N, dev_AA, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);
    
    // print the result
    
    for (int i = 0; i < parasize; i++){
        printf("thread %d move A=%s to N=%s\n",i,Put(A,HEX),Put(h_N[i],HEX));
    }

    Mov_Big_Big(N,h_N[0]);
    
    // free the device yo
    hipFree(h_N);
    hipFree(dev_N);
    hipFree(h_AA);
    hipFree(dev_AA);

}


