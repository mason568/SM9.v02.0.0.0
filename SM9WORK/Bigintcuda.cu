#include "hip/hip_runtime.h"
#include "Bigintcuda.cuh"

__device__ int dev_cmp(CBigInt *d_N,CBigInt *d_A)
{
	int i;
	if(d_N->m_nLength>d_A->m_nLength)
		return 1;
	if(d_N->m_nLength<d_A->m_nLength)
		return -1;
	for(i=d_N->m_nLength-1;i>=0;i--)
	{
		if(d_N->m_ulValue[i]>d_A->m_ulValue[i])
			return 1;
		if(d_N->m_ulValue[i]<d_A->m_ulValue[i])
			return -1;
	}
	return 0;
}

__device__ void dev_mov_big_long(CBigInt *d_N,unsigned long d_A){
    int i;
	if(d_A>0xffffffff)
	{
		d_N->m_nLength=2;
		d_N->m_ulValue[1]=(unsigned long)((unsigned long long)d_A>>32);
		d_N->m_ulValue[0]=(unsigned long)d_A;
	}
	else
	{
		d_N->m_nLength=1;
		d_N->m_ulValue[0]=(unsigned long)d_A;
	}
	for(i=d_N->m_nLength;i<BI_MAXLEN;i++)
	{
		d_N->m_ulValue[i]=0;
	}
}

//构造大数对象并初始化为零 + 其对应的kernal函数

__global__ void cbiginiinit_thread(CBigInt *d_out){
    int idx = threadIdx.x; // this is how you get the thread index
	d_out[idx].m_nLength=1;
	for(int i=0;i<BI_MAXLEN;i++)
		d_out[idx].m_ulValue[i]=0;
}

void CBigIntInit_para(CBigInt *A,int parasize)
{
    CBigInt *h_A, *dev_A;


    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_A,parasize*(sizeof(CBigInt)),hipHostMallocDefault));

    
	CHECK(hipMalloc((void **)&dev_A,parasize*(sizeof(CBigInt))));
    // copy from host memory to pinned memory

    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    //hipMemcpy(dev_A, h_A, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    cbiginiinit_thread<<<1,parasize>>>(dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_A, dev_A, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    // print the result
    
    for (int i = 0; i < parasize; i++){
        printf("para %d = %s\n",i,Put(h_A[i],HEX));
    }

    Mov_Big_Big(A,h_A[0]);

    // free the device yo
    hipFree(h_A);
    hipFree(dev_A);
}


/****************************************************************************************
大数比较
调用方式：Cmp_para(N,A,num)
返回值：若N<A返回-1；若N=A返回0；若N>A返回1

****************************************************************************************/
__global__ void cmp_thread(CBigInt *dev_N,CBigInt *dev_A, int *d_cmp_res){
    int idx = threadIdx.x; // this is how you get the thread index



	int i;
	if(dev_N[idx].m_nLength>dev_A[idx].m_nLength){
		d_cmp_res[idx] =1;
        return;
    }
	if(dev_N[idx].m_nLength<dev_A[idx].m_nLength){
		d_cmp_res[idx] =-1;
        return;
    }
	for(i=dev_N[idx].m_nLength-1;i>=0;i--)
	{
		if(dev_N[idx].m_ulValue[i]>dev_A[idx].m_ulValue[i]){
            d_cmp_res[idx] =1;
            return;
        }
		if(dev_N[idx].m_ulValue[i]<dev_A[idx].m_ulValue[i]){
            d_cmp_res[idx] =-1;
            return;
        }
	}
    d_cmp_res[idx] = 0;
	return;
}

int Cmp_para(CBigInt N,CBigInt A, int parasize)
{

    CBigInt *h_N, *h_A, *dev_N,*dev_A;
    int *h_cmp_ret, *d_cmp_ret;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    for(int i=0;i<parasize;i++){
        Mov_Big_Big(&h_N[i],N);
        Mov_Big_Big(&h_A[i],A);
    }
    CHECK(hipHostAlloc((void**) &h_cmp_ret,parasize*(sizeof(int)),hipHostMallocDefault));
    CHECK(hipMalloc((void **)&dev_N,parasize*(sizeof(CBigInt))));
    CHECK(hipMalloc((void **)&dev_A,parasize*(sizeof(CBigInt))));
    CHECK(hipMalloc((void **)&d_cmp_ret,parasize*(sizeof(int))));
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    cmp_thread<<<1,parasize>>>(dev_N,dev_A,d_cmp_ret);

    // copy the result back to the CPU mem
    hipMemcpy(h_cmp_ret, d_cmp_ret, parasize*(sizeof(int)), hipMemcpyDeviceToHost);
    for(int i=0;i<parasize;i++){
        printf("para %d cmp_ret = %d\n",i, h_cmp_ret[i]);
    }
    return h_cmp_ret[0];
}

/****************************************************************************************
大数赋值
调用方式：assignn_Big_to_Big_para(&N,A,32)  
assign_Long_to_Big_para(&N,A,32)
返回值：多个N,被赋值为同一个A
但是嘛，由于cpu和gpu之间数据交换也需要时间，开销不好说，暂时写了，但可以不用
****************************************************************************************/
__global__ void mov_big_big_thread(CBigInt *dev_N, CBigInt *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
    dev_N[idx].m_nLength=dev_A[idx].m_nLength;

	for(int i=0;i<BI_MAXLEN;i++)
	{
		dev_N[idx].m_ulValue[i]=dev_A[idx].m_ulValue[i];
	}
}
__global__ void assignn_Big_to_Big_thread(CBigInt *dev_N, CBigInt *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
    dev_N[idx].m_nLength=dev_A->m_nLength;

	for(int i=0;i<BI_MAXLEN;i++)
	{
		dev_N[idx].m_ulValue[i]=dev_A->m_ulValue[i];
	}
}
__global__ void assignn_Long_to_Big_thread(CBigInt *dev_N, unsigned long *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
	int i;
	if(*dev_A > (unsigned long)0xffffffff)
	{
		dev_N[idx].m_nLength=2;
		dev_N[idx].m_ulValue[1]=(unsigned long)((unsigned long long)*dev_A>>32);
		dev_N[idx].m_ulValue[0]=(unsigned long)*dev_A;
	}
	else
	{
		dev_N[idx].m_nLength=1;
		dev_N[idx].m_ulValue[0]=(unsigned long)*dev_A;
	}
	for(i=dev_N[idx].m_nLength;i<BI_MAXLEN;i++)
	{
		dev_N[idx].m_ulValue[i]=0;
	}
}
//一个Big为parasize个Big赋值  A ->  parasize*N
void assignn_Big_to_Big_para(CBigInt *N, CBigInt A, int parasize)
{
    CBigInt *h_N, *dev_N;
    CBigInt *h_A,*dev_A;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    //CHECK(hipHostAlloc((void**) &h_A,sizeof(int),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));
    Mov_Big_Big(h_A,A);
    CHECK(hipMalloc((void **)&dev_N,parasize*(sizeof(CBigInt))));
    //CHECK(hipMalloc((void **)&dev_A,parasize*(sizeof(CBigInt))));
	CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    // copy from host memory to pinned memory
    
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(CBigInt), hipMemcpyHostToDevice);

    // launch the kernel
    assignn_Big_to_Big_thread<<<1,parasize>>>(dev_N,dev_A);
    
    // copy the result back to the CPU mem
    hipMemcpy(h_N, dev_N, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);
   
    // print the result
    
    for (int i = 0; i < parasize; i++){
        printf("thread %d move A=%s to N=%s\n",i,Put(A,HEX),Put(h_N[i],HEX));
    }
    memcpy(N,h_N,parasize*(sizeof(CBigInt)));
    //Mov_Big_Big(N,h_N[0]);
   
    // free the device yo
    hipFree(h_N);
    hipFree(dev_N);
    hipFree(h_A);
    hipFree(dev_A);

}
//一个unsigned long为parasize个Big赋值
void assignn_Long_to_Big_para(CBigInt *N, unsigned long A, int parasize)
{
    CBigInt *h_N, *dev_N;
    unsigned long *h_A,*dev_A;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    //CHECK(hipHostAlloc((void**) &h_A,sizeof(int),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(unsigned long),hipHostMallocDefault));
    //Mov_Big_Long(h_A,A);
    memcpy(h_A,&A,sizeof(unsigned long));
    
    CHECK(hipMalloc((void **)&dev_N,parasize*(sizeof(CBigInt))));
    //CHECK(hipMalloc((void **)&dev_A,parasize*(sizeof(CBigInt))));
	CHECK(hipMalloc((void **)&dev_A,sizeof(unsigned long)));
    // copy from host memory to pinned memory
    
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(unsigned long), hipMemcpyHostToDevice);

    // launch the kernel
    assignn_Long_to_Big_thread<<<1,parasize>>>(dev_N,dev_A);
    
    // copy the result back to the CPU mem
    hipMemcpy(h_N, dev_N, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);
   
    // print the result
    
    for (int i = 0; i < parasize; i++){
        printf("thread %d move A=%ld to N=%s\n",i,A,Put(h_N[i],HEX));
    }
    memcpy(N,h_N,parasize*(sizeof(CBigInt)));
    //Mov_Big_Big(N,h_N[0]);
    // free the device yo
    hipFree(h_N);
    hipFree(dev_N);
    hipFree(h_A);
    hipFree(dev_A);

}


void Mov_Big_Big_para(CBigInt *N, CBigInt A, int parasize)
{
    CBigInt *h_N, *dev_N, *h_AA, *dev_AA;
    //CBigInt h_A,dev_A;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_AA,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    for(int i=0;i<parasize;i++){
        Mov_Big_Big(&h_AA[i],A);
    }
    //CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));

    CHECK(hipMalloc((void **)&dev_N,parasize*(sizeof(CBigInt))));
    CHECK(hipMalloc((void **)&dev_AA,parasize*(sizeof(CBigInt))));
	//CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    // copy from host memory to pinned memory
    
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_AA, h_AA, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    mov_big_big_thread<<<1,parasize>>>(dev_N,dev_AA);
   
    // copy the result back to the CPU mem
    hipMemcpy(h_N, dev_AA, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);
    
    // print the result
    
    for (int i = 0; i < parasize; i++){
        printf("thread %d move A=%s to N=%s\n",i,Put(A,HEX),Put(h_N[i],HEX));
    }

    Mov_Big_Big(N,h_N[0]);
    
    // free the device yo
    hipFree(h_N);
    hipFree(dev_N);
    hipFree(h_AA);
    hipFree(dev_AA);

}


/****************************************************************************************
大数相加
调用形式：Add_Big_Big_para(Y,N,A,num)  Add_Big_Long_para(Y,N,A,num)
返回值：Y=N+A
****************************************************************************************/
__global__ void add_big_big_thread(CBigInt *dev_YY, CBigInt *dev_N,CBigInt *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
    //直接在dev_YY[idx]上操作
    unsigned int i;
	//CBigInt X;
	unsigned int carry=0;
	unsigned long long sum=0;
	//CBigIntInit(&X);
	//Mov_Big_Big(&(dev_YY[idx]), *dev_N);
	if(dev_YY[idx].m_nLength<dev_A->m_nLength)
	{
		dev_YY[idx].m_nLength=dev_A->m_nLength;
	}
	for(i=0;i<dev_YY[idx].m_nLength;i++)
	{
		sum=dev_A->m_ulValue[i];
		sum=sum+dev_YY[idx].m_ulValue[i]+carry;
		dev_YY[idx].m_ulValue[i]=(unsigned long)sum;
		carry=(unsigned int)(sum>>32);
	}
	dev_YY[idx].m_ulValue[dev_YY[idx].m_nLength]=carry;
	dev_YY[idx].m_nLength+=carry;	
}


void Add_Big_Big_para(CBigInt *Y, CBigInt N, CBigInt A, int parasize)
{
    CBigInt *h_N, *h_A, *h_YY, *dev_N,*dev_A,*dev_YY;
    
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_YY,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(CBigInt));
    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_YY,parasize*(sizeof(CBigInt))));
    assignn_Big_to_Big_para(h_YY,N,32);
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_YY, h_YY, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);
    // launch the kernel
    add_big_big_thread<<<1,parasize>>>(dev_YY,dev_N,dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_YY, dev_YY, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);
    for(int i=0;i<parasize;i++){
        printf("h_YY[%d] = %s\n", i,Put(h_YY[i],HEX));
    }
    

    Mov_Big_Big(Y,h_YY[0]);
    // free the device yo
    hipFree(h_N);
    hipFree(h_A);
    hipFree(h_YY);
    hipFree(dev_N);
    hipFree(dev_A);
    hipFree(dev_YY);
}


__global__ void add_big_long_thread(CBigInt *dev_YY, CBigInt *dev_N,unsigned long *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
    //直接在dev_YY[idx]上操作
	unsigned long sum;
	sum=dev_YY[idx].m_ulValue[0];
	sum+=(unsigned long)*dev_A;     
	dev_YY[idx].m_ulValue[0]=sum;
	if(sum>(unsigned long)0xffffffff)
	{
		unsigned int i=1;
		while(dev_YY[idx].m_ulValue[i]==0xffffffff)
		{
			dev_YY[idx].m_ulValue[i]=0;
			i++;
		}
		dev_YY[idx].m_ulValue[i]++;
		if(dev_N->m_nLength==i)
			dev_N->m_nLength++;
	}
	
}


void Add_Big_Long_para(CBigInt *Y, CBigInt N, unsigned long A, int parasize)
{
	CBigInt *h_N, *h_YY, *dev_N,*dev_YY;
    unsigned long *h_A, *dev_A;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(unsigned long),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_YY,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(unsigned long));
    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(unsigned long)));
    CHECK(hipMalloc((void **)&dev_YY,parasize*(sizeof(CBigInt))));
    //printf("ok4\n");
    assignn_Big_to_Big_para(h_YY,N,32);
    //printf("ok5\n");
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(unsigned long), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_YY, h_YY, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);
    for(int i=0;i<parasize;i++){
        printf("h_YY[%d] = %s\n", i,Put(h_YY[i],HEX));
    }
    printf("ok6\n");
    // launch the kernel
    add_big_long_thread<<<1,parasize>>>(dev_YY,dev_N,dev_A);
    // copy the result back to the CPU mem
    hipMemcpy(h_YY, dev_YY, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);
    for(int i=0;i<parasize;i++){
        printf("h_YY[%d] = %s\n", i,Put(h_YY[i],HEX));
    }
    Mov_Big_Big(Y,h_YY[0]);
    // free the device yo
    hipFree(h_N);
    hipFree(h_A);
    hipFree(h_YY);
    hipFree(dev_N);
    hipFree(dev_A);
    hipFree(dev_YY);
    
}


/****************************************************************************************
大数相减
调用形式：Sub_Big_Big_para(Y,N,A,num) Sub_Big_Long_para(Y,N,A,num)
返回值：Y=N-A
****************************************************************************************/
__global__ void sub_big_big_thread(CBigInt *dev_YY, CBigInt *dev_N,CBigInt *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index

	unsigned int carry=0;
	unsigned long long num;
	unsigned int i;

	if(dev_cmp(&(dev_YY[idx]),dev_A)<=0)
		dev_mov_big_long(&dev_YY[idx],0);
    else
	{
		for(i=0;i<dev_N->m_nLength;i++)
		{
			if((dev_N->m_ulValue[i]>dev_YY[idx].m_ulValue[i])||((dev_N->m_ulValue[i]==dev_YY[idx].m_ulValue[i])&&(carry==0)))
			{
				dev_YY[idx].m_ulValue[i]=dev_N->m_ulValue[i]-carry-dev_A->m_ulValue[i];
				carry=0;
			}
			else
			{
				num=0x100000000+dev_YY[idx].m_ulValue[i];
				dev_YY[idx].m_ulValue[i]=(unsigned long)(num-carry-dev_YY[idx].m_ulValue[i]);
				carry=1;
			}     
		}
		while(dev_YY[idx].m_ulValue[dev_YY[idx].m_nLength-1]==0)
			dev_YY[idx].m_nLength--;
			
	}	
}

void Sub_Big_Big_para(CBigInt *Y, CBigInt N, CBigInt A, int parasize)
{
    CBigInt *h_N, *h_A, *h_YY, *dev_N,*dev_A,*dev_YY;
    
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_YY,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    //printf("ok2\n");
    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(CBigInt));
    //printf("h_N = %s\n",Put(*h_N,HEX));
    //printf("h_A = %s\n",Put(*h_A,HEX));
    //printf("ok3\n");
    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_YY,parasize*(sizeof(CBigInt))));
    //printf("ok4\n");
    assignn_Big_to_Big_para(h_YY,N,32);
    //printf("ok5\n");
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_YY, h_YY, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);
    printf("ok6\n");
    // launch the kernel
    sub_big_big_thread<<<1,parasize>>>(dev_YY,dev_N,dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_YY, dev_YY, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    for(int i=0;i<parasize;i++){
        printf("h_YY[%d] = %s\n", i,Put(h_YY[i],HEX));
    }

    Mov_Big_Big(Y,h_YY[0]);
    // free the device yo
    hipFree(h_N);
    hipFree(h_A);
    hipFree(h_YY);
    hipFree(dev_N);
    hipFree(dev_A);
    hipFree(dev_YY);
}