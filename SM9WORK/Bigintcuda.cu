#include "hip/hip_runtime.h"
#include "Bigintcuda.cuh"


__device__ int dev_cmp(CBigInt *d_N,CBigInt *d_A)
{
	int i;
	if(d_N->m_nLength>d_A->m_nLength)
		return 1;
	if(d_N->m_nLength<d_A->m_nLength)
		return -1;
	for(i=d_N->m_nLength-1;i>=0;i--)
	{
		if(d_N->m_ulValue[i]>d_A->m_ulValue[i])
			return 1;
		if(d_N->m_ulValue[i]<d_A->m_ulValue[i])
			return -1;
	}
	return 0;
}

__device__ void dev_mov_big_long(CBigInt *d_N,unsigned long d_A){
    int i;
	if(d_A>0xffffffff)
	{
		d_N->m_nLength=2;
		d_N->m_ulValue[1]=(unsigned long)((unsigned long long)d_A>>32);
		d_N->m_ulValue[0]=(unsigned long)d_A;
	}
	else
	{
		d_N->m_nLength=1;
		d_N->m_ulValue[0]=(unsigned long)d_A;
	}
	for(i=d_N->m_nLength;i<BI_MAXLEN;i++)
	{
		d_N->m_ulValue[i]=0;
	}
}

__device__ void dev_mov_big_big(CBigInt *d_N, CBigInt d_A)
{
	//CBigInt N;
	int i;
	d_N->m_nLength=d_A.m_nLength;
	for(i=0;i<BI_MAXLEN;i++)
	{
		d_N->m_ulValue[i]=d_A.m_ulValue[i];
	}
	
}

__device__ void dev_cbigintinit(CBigInt *d_A)
{
	int i;
	d_A->m_nLength=1;
	for(i=0;i<BI_MAXLEN;i++)
		d_A->m_ulValue[i]=0;
}

__device__ void dev_add_big_long(CBigInt *Y, CBigInt N, unsigned long A)//
{
	CBigInt X;
	unsigned long long sum;
	dev_cbigintinit(&X);
	dev_mov_big_big(&X,N);// X=N

	sum=X.m_ulValue[0];
	sum+=A;     
	X.m_ulValue[0]=(unsigned long)sum;
	if(sum>0xffffffff)
	{
		unsigned int i=1;
		while(X.m_ulValue[i]==0xffffffff)
		{
			X.m_ulValue[i]=0;
			i++;
		}
		X.m_ulValue[i]++;
		if(N.m_nLength==i)
			N.m_nLength++;
	}
	dev_mov_big_big(Y,X);
}

__device__ void dev_add_big_big(CBigInt *Y, CBigInt N, CBigInt A)
{
	unsigned int i;
	CBigInt X;
	unsigned int carry=0;
	unsigned long long sum=0;
	dev_cbigintinit(&X);
	dev_mov_big_big(&X, N);

	if(X.m_nLength<A.m_nLength)
	{
		X.m_nLength=A.m_nLength;
	}
	for(i=0;i<X.m_nLength;i++)
	{
		sum=A.m_ulValue[i];
		sum=sum+X.m_ulValue[i]+carry;
		X.m_ulValue[i]=(unsigned long)sum;
		carry=(unsigned int)(sum>>32);
	}
	X.m_ulValue[X.m_nLength]=carry;
	X.m_nLength+=carry;	
	dev_mov_big_big(Y,X);
}

__device__ void dev_mul_big_long(CBigInt *Y, CBigInt N, unsigned long A)
{
	CBigInt X;
	unsigned long long mul;
	unsigned long carry=0;
	unsigned int i;
	dev_cbigintinit(&X);
	dev_mov_big_big(&X,N);;
	for(i=0;i<N.m_nLength;i++)
	{
		mul=N.m_ulValue[i];
		mul=mul*A+carry;
		X.m_ulValue[i]=(unsigned int)mul;
		carry=(unsigned int)(mul>>32);
	}
	if(carry)
	{
		X.m_nLength++;
		X.m_ulValue[X.m_nLength-1]=carry;
	}
	i = X.m_nLength-1;
	while(X.m_ulValue[i]==0 && i>0)
	{
		X.m_nLength--;
		i--;
	}
	dev_mov_big_big(Y,X);
}

__device__ void dev_mul_big_big(CBigInt *Y, CBigInt N, CBigInt A)
{
	CBigInt X;
	unsigned long long sum,mul=0,carry=0;
	unsigned int i,j;
	if(A.m_nLength==1)
		dev_mul_big_long(Y,N,A.m_ulValue[0]);
	else
	{
		dev_cbigintinit(&X);
		X.m_nLength=N.m_nLength+A.m_nLength-1;
		for(i=0;i<X.m_nLength;i++)
		{
			sum=carry;
			carry=0;
			for(j=0;j<A.m_nLength;j++)
			{
				if(((i-j)>=0)&&((i-j)<N.m_nLength))
				{
					mul=N.m_ulValue[i-j];
					mul*=A.m_ulValue[j];
					carry+=mul>>32;
					mul=mul&0xffffffff;
					sum+=mul;
				}
			}				
			carry+=sum>>32;
			X.m_ulValue[i]=(unsigned long)sum;
		}
		if(carry)
		{
			X.m_nLength++;
			X.m_ulValue[X.m_nLength-1]=(unsigned long)carry;
		}
		i = X.m_nLength-1;
		while(X.m_ulValue[i]==0 && i>0)
		{
			X.m_nLength--;
			i--;
		}
		dev_mov_big_big(Y,X);
	}
}

__device__ void dev_sub_big_big(CBigInt *Y, CBigInt N, CBigInt A)
{
	CBigInt X;
	unsigned int carry=0;
	unsigned long long num;
	unsigned int i;
	dev_cbigintinit(&X);
	dev_mov_big_big(&X,N);
	if(dev_cmp(&X,&A)<=0)
		dev_mov_big_long(&X,0);
    else
	{

		for(i=0;i<N.m_nLength;i++)
		{
			if((N.m_ulValue[i]>A.m_ulValue[i])||((N.m_ulValue[i]==A.m_ulValue[i])&&(carry==0)))
			{
				X.m_ulValue[i]=N.m_ulValue[i]-carry-A.m_ulValue[i];
				carry=0;
			}
			else
			{
				num=0x100000000+X.m_ulValue[i];
				X.m_ulValue[i]=(unsigned long)(num-carry-A.m_ulValue[i]);
				carry=1;
			}     
		}
		while(X.m_ulValue[X.m_nLength-1]==0)
			X.m_nLength--;
			
	}
	dev_mov_big_big(Y,X);
}


__device__ unsigned long dev_mod_big_long(CBigInt N, unsigned long A)
{
	unsigned long long div;
	unsigned long carry=0;
	int i;
	if(N.m_nLength==1)
		return(N.m_ulValue[0]%A);


	for(i=N.m_nLength-1;i>=0;i--)
	{
		div=N.m_ulValue[i];
		div+=carry*0x100000000;
		carry=(unsigned long)(div%A);
	}
	return carry;
}


__device__ void dev_mod_big_big(CBigInt *Z,CBigInt N, CBigInt A)
{
	CBigInt X,Y;
	unsigned long long div,num;
	//unsigned long carry=0;
	unsigned int i,len;
	dev_cbigintinit(&X);
	dev_cbigintinit(&Y);
	dev_mov_big_big(&X,N);//dev_mov_big_big
	while(dev_cmp(&X,&A)>=0)
	{
		div=X.m_ulValue[X.m_nLength-1];
		num=A.m_ulValue[A.m_nLength-1];
		len=X.m_nLength-A.m_nLength;
		if((div==num)&&(len==0))
		{
			//Mov_Big_Big(&X,Sub_Big_Big(X,A));
			dev_sub_big_big(&X,X,A);//dev_sub_big_big
			break;
		}
		if((div<=num)&&len)
		{
			len--;div=(div<<32)+X.m_ulValue[X.m_nLength-2];
		}
		div=div/(num+1);
		dev_mov_big_long(&Y,div);	//dev_mov_big_long	
		//Mov_Big_Big(&Y,Mul_Big_Big(A,Y));
		dev_mul_big_big(&Y,A,Y); //dev_mul_big_big
		if(len)
		{
			Y.m_nLength+=len;
			for(i=Y.m_nLength-1;i>=len;i--)
				Y.m_ulValue[i]=Y.m_ulValue[i-len];
			for(i=0;i<len;i++)
				Y.m_ulValue[i]=0;
		}
		//Mov_Big_Big(&X,Sub_Big_Big(X,Y));
		dev_sub_big_big(&X,X,Y);
	}
	dev_mov_big_big(Z,X);
}

__device__ void dev_div_big_long(CBigInt *Y, CBigInt N, unsigned long A)
{
	CBigInt X;
	unsigned long long div,mul;
	unsigned long carry=0;
	int i;
	//CBigIntInit(X);
	dev_mov_big_big(&X,N);;
	if(X.m_nLength==1)
	{
		X.m_ulValue[0]=X.m_ulValue[0]/A;
		dev_mov_big_big(Y,X);
	}
	else
	{
		for(i=X.m_nLength-1;i>=0;i--)
		{
			div=carry;
			div=(div<<32)+X.m_ulValue[i];
			X.m_ulValue[i]=(unsigned long)(div/A);
			mul=(div/A)*A;
			carry=(unsigned long)(div-mul);
		}
		if(X.m_ulValue[X.m_nLength-1]==0)
			X.m_nLength--;
		dev_mov_big_big(Y,X);
	}
}

__device__ void dev_div_big_big(CBigInt *M,CBigInt N, CBigInt A)
{
	CBigInt X,Y,Z,T;
	unsigned int i,len;
	unsigned long long num,div;
	if(A.m_nLength==1)
		dev_div_big_long(M,N,A.m_ulValue[0]);
	else
	{
		dev_cbigintinit(&X);
		dev_cbigintinit(&Y);
		dev_cbigintinit(&Z);
		dev_cbigintinit(&T);
		dev_mov_big_big(&Y,N);
		while(dev_cmp(&Y,&A)>=0)
		{        
			div=Y.m_ulValue[Y.m_nLength-1];
			num=A.m_ulValue[A.m_nLength-1];
			len=Y.m_nLength-A.m_nLength;
			if((div==num)&&(len==0))
			{
				//Mov_Big_Big(&X,Add_Big_Long(X,1));
				dev_add_big_long(&X,X,1);//dev_add_big_long
				break;
			}
			if((div<=num)&&len)
			{
				len--;
				div=(div<<32)+Y.m_ulValue[Y.m_nLength-2];
			}
			div=div/(num+1);
			dev_mov_big_long(&Z,div);
			if(len)
			{
				Z.m_nLength+=len;
				for(i=Z.m_nLength-1;i>=len;i--)
					Z.m_ulValue[i]=Z.m_ulValue[i-len];
				for(i=0;i<len;i++)
					Z.m_ulValue[i]=0;
			}
			//Mov_Big_Big(&X,Add_Big_Big(X,Z));
			dev_add_big_big(&X,X,Z);
			//Mov_Big_Big(&Y,Sub_Big_Big(Y,Mul_Big_Big(A,Z)));
			dev_mul_big_big(&T,A,Z);
			dev_sub_big_big(&Y,Y,T);
		}
		dev_mov_big_big(M,X);
	}
}


__device__ char* dev_put(CBigInt *N, unsigned int system)
{
	char* str;	
	char t[17]="0123456789ABCDEF";
	int i, a, len;
	char s[1024] = {"0"};
	char s1[1024] = {"0"};
	CBigInt X;
	str="";
	if((N->m_nLength==1)&&(N->m_ulValue[0]==0))
	{
		str="0";
		return str;
	}

	dev_cbigintinit(&X);
	dev_mov_big_big(&X,*N);;
	for(i = 0; X.m_ulValue[X.m_nLength-1]>0; i ++)
	{
		a=dev_mod_big_long(X,system);
		s[i] = t[a]; 
		//Mov_Big_Big(&X,Div_Big_Long(X,system));
		dev_div_big_long(&X,X,system);
	}
	len = i-1;  
	for( i= 0; i<= len; i++)
		s1[i] = s[len-i];
	str = s1;
	return str;
}
/****************************************************************************************
//构造大数对象并初始化为零 + 其对应的kernal函数
若想返回parasize的数据量请启用 free the device yo 之前的memcpy 并注释替代函数
****************************************************************************************/
__global__ void cbiginiinit_thread(CBigInt *d_out){
    int idx = threadIdx.x; // this is how you get the thread index
	d_out[idx].m_nLength=1;
	for(int i=0;i<BI_MAXLEN;i++)
		d_out[idx].m_ulValue[i]=0;
}

void CBigIntInit_para(CBigInt *A,int parasize)
{
    CBigInt *h_A, *dev_A;


    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_A,parasize*(sizeof(CBigInt)),hipHostMallocDefault));

    
	CHECK(hipMalloc((void **)&dev_A,parasize*(sizeof(CBigInt))));
    // copy from host memory to pinned memory

    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    //hipMemcpy(dev_A, h_A, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    cbiginiinit_thread<<<1,parasize>>>(dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_A, dev_A, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    // print the result
    
    for (int i = 0; i < parasize; i++){
        printf("para %d = %s\n",i,Put(h_A[i],HEX));
    }

    Mov_Big_Big(A,h_A[0]);
    //memcpy(A,h_A,parasize*(sizeof(CBigInt)));
    // free the device yo
    hipFree(h_A);
    hipFree(dev_A);
}


/****************************************************************************************
大数比较
调用方式：Cmp_para(N,A,num)
返回值：若N<A返回-1；若N=A返回0；若N>A返回1
这个函数目前没有返回一个链表，还是返回一个数
****************************************************************************************/
__global__ void cmp_thread(CBigInt *dev_N,CBigInt *dev_A, int *d_cmp_res){
    int idx = threadIdx.x; // this is how you get the thread index



	int i;
	if(dev_N[idx].m_nLength>dev_A[idx].m_nLength){
		d_cmp_res[idx] =1;
        return;
    }
	if(dev_N[idx].m_nLength<dev_A[idx].m_nLength){
		d_cmp_res[idx] =-1;
        return;
    }
	for(i=dev_N[idx].m_nLength-1;i>=0;i--)
	{
		if(dev_N[idx].m_ulValue[i]>dev_A[idx].m_ulValue[i]){
            d_cmp_res[idx] =1;
            return;
        }
		if(dev_N[idx].m_ulValue[i]<dev_A[idx].m_ulValue[i]){
            d_cmp_res[idx] =-1;
            return;
        }
	}
    d_cmp_res[idx] = 0;
	return;
}

int Cmp_para(CBigInt N,CBigInt A, int parasize)
{
    
    CBigInt *h_N, *h_A, *dev_N,*dev_A;
    int *h_cmp_ret, *d_cmp_ret;
    int ret;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    for(int i=0;i<parasize;i++){
        Mov_Big_Big(&h_N[i],N);
        Mov_Big_Big(&h_A[i],A);
    }
    CHECK(hipHostAlloc((void**) &h_cmp_ret,parasize*(sizeof(int)),hipHostMallocDefault));
    CHECK(hipMalloc((void **)&dev_N,parasize*(sizeof(CBigInt))));
    CHECK(hipMalloc((void **)&dev_A,parasize*(sizeof(CBigInt))));
    CHECK(hipMalloc((void **)&d_cmp_ret,parasize*(sizeof(int))));
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    cmp_thread<<<1,parasize>>>(dev_N,dev_A,d_cmp_ret);

    // copy the result back to the CPU mem
    hipMemcpy(h_cmp_ret, d_cmp_ret, parasize*(sizeof(int)), hipMemcpyDeviceToHost);
    for(int i=0;i<parasize;i++){
        printf("para %d cmp_ret = %d\n",i, h_cmp_ret[i]);
    }
    //memcpy(N,h_N,parasize*(sizeof(CBigInt)));  
    ret = h_cmp_ret[0];
        // free the device yo
    hipFree(h_N);
    hipFree(dev_N);
    hipFree(h_A);
    hipFree(dev_A);
    hipFree(h_cmp_ret);
    hipFree(d_cmp_ret);
    return ret;
}

/****************************************************************************************
大数赋值
调用方式：assignn_Big_to_Big_para(&N,A,32)  
assign_Long_to_Big_para(&N,A,32)

若想返回parasize的数据量请启用 free the device yo 之前的memcpy 并注释替代函数
****************************************************************************************/
__global__ void mov_big_big_thread(CBigInt *dev_N, CBigInt *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
    dev_N[idx].m_nLength=dev_A[idx].m_nLength;

	for(int i=0;i<BI_MAXLEN;i++)
	{
		dev_N[idx].m_ulValue[i]=dev_A[idx].m_ulValue[i];
	}
}
__global__ void assignn_Big_to_Big_thread(CBigInt *dev_N, CBigInt *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
    dev_N[idx].m_nLength=dev_A->m_nLength;

	for(int i=0;i<BI_MAXLEN;i++)
	{
		dev_N[idx].m_ulValue[i]=dev_A->m_ulValue[i];
	}
}
__global__ void assignn_Long_to_Big_thread(CBigInt *dev_N, unsigned long *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
	int i;
	if(*dev_A > (unsigned long)0xffffffff)
	{
		dev_N[idx].m_nLength=2;
		dev_N[idx].m_ulValue[1]=(unsigned long)((unsigned long long)*dev_A>>32);
		dev_N[idx].m_ulValue[0]=(unsigned long)*dev_A;
	}
	else
	{
		dev_N[idx].m_nLength=1;
		dev_N[idx].m_ulValue[0]=(unsigned long)*dev_A;
	}
	for(i=dev_N[idx].m_nLength;i<BI_MAXLEN;i++)
	{
		dev_N[idx].m_ulValue[i]=0;
	}
}
//一个Big为parasize个Big赋值  A ->  parasize*N
void assignn_Big_to_Big_para(CBigInt *N, CBigInt A, int parasize)
{
    CBigInt *h_N, *dev_N;
    CBigInt *h_A,*dev_A;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    //CHECK(hipHostAlloc((void**) &h_A,sizeof(int),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));
    Mov_Big_Big(h_A,A);
    CHECK(hipMalloc((void **)&dev_N,parasize*(sizeof(CBigInt))));
    //CHECK(hipMalloc((void **)&dev_A,parasize*(sizeof(CBigInt))));
	CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    // copy from host memory to pinned memory
    
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(CBigInt), hipMemcpyHostToDevice);

    // launch the kernel
    assignn_Big_to_Big_thread<<<1,parasize>>>(dev_N,dev_A);
    
    // copy the result back to the CPU mem
    hipMemcpy(h_N, dev_N, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);
   
    // print the result
    /*
    for (int i = 0; i < parasize; i++){
        printf("thread %d move A=%s to N=%s\n",i,Put(A,HEX),Put(h_N[i],HEX));
    }
    */
    memcpy(N,h_N,parasize*(sizeof(CBigInt)));
    //Mov_Big_Big(N,h_N[0]);
   
    // free the device yo
    hipFree(h_N);
    hipFree(dev_N);
    hipFree(h_A);
    hipFree(dev_A);

}
//一个unsigned long为parasize个Big赋值
void assignn_Long_to_Big_para(CBigInt *N, unsigned long A, int parasize)
{
    CBigInt *h_N, *dev_N;
    unsigned long *h_A,*dev_A;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    //CHECK(hipHostAlloc((void**) &h_A,sizeof(int),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(unsigned long),hipHostMallocDefault));
    //Mov_Big_Long(h_A,A);
    memcpy(h_A,&A,sizeof(unsigned long));
    
    CHECK(hipMalloc((void **)&dev_N,parasize*(sizeof(CBigInt))));
    //CHECK(hipMalloc((void **)&dev_A,parasize*(sizeof(CBigInt))));
	CHECK(hipMalloc((void **)&dev_A,sizeof(unsigned long)));
    // copy from host memory to pinned memory
    
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(unsigned long), hipMemcpyHostToDevice);

    // launch the kernel
    assignn_Long_to_Big_thread<<<1,parasize>>>(dev_N,dev_A);
    
    // copy the result back to the CPU mem
    hipMemcpy(h_N, dev_N, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);
   
    // print the result
    /*
    for (int i = 0; i < parasize; i++){
        printf("thread %d move A=%ld to N=%s\n",i,A,Put(h_N[i],HEX));
    }
    */
    memcpy(N,h_N,parasize*(sizeof(CBigInt)));
    //Mov_Big_Big(N,h_N[0]);
    // free the device yo
    hipFree(h_N);
    hipFree(dev_N);
    hipFree(h_A);
    hipFree(dev_A);

}


void Mov_Big_Big_para(CBigInt *N, CBigInt A, int parasize)
{
    CBigInt *h_N, *dev_N, *h_AA, *dev_AA;
    //CBigInt h_A,dev_A;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_AA,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    for(int i=0;i<parasize;i++){
        Mov_Big_Big(&h_AA[i],A);
    }
    //CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));

    CHECK(hipMalloc((void **)&dev_N,parasize*(sizeof(CBigInt))));
    CHECK(hipMalloc((void **)&dev_AA,parasize*(sizeof(CBigInt))));
	//CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    // copy from host memory to pinned memory
    
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_AA, h_AA, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    mov_big_big_thread<<<1,parasize>>>(dev_N,dev_AA);
   
    // copy the result back to the CPU mem
    hipMemcpy(h_N, dev_AA, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);
    
    // print the result
    /*
    for (int i = 0; i < parasize; i++){
        printf("thread %d move A=%s to N=%s\n",i,Put(A,HEX),Put(h_N[i],HEX));
    }
    */

    Mov_Big_Big(N,h_N[0]);
    //memcpy(N,h_N,parasize*(sizeof(CBigInt)));
    // free the device yo
    hipFree(h_N);
    hipFree(dev_N);
    hipFree(h_AA);
    hipFree(dev_AA);

}


/****************************************************************************************
大数相加
调用形式：Add_Big_Big_para(Y,N,A,num)  Add_Big_Long_para(Y,N,A,num)
返回值：Y=N+A
若想返回parasize的数据量请启用 free the device yo 之前的memcpy 并注释替代函数
****************************************************************************************/
__global__ void add_big_big_thread(CBigInt *dev_YY, CBigInt *dev_N,CBigInt *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
    //直接在dev_YY[idx]上操作
    unsigned int i;
	//CBigInt X;
	unsigned int carry=0;
	unsigned long long sum=0;
	//CBigIntInit(&X);
	//Mov_Big_Big(&(dev_YY[idx]), *dev_N);
	if(dev_YY[idx].m_nLength<dev_A->m_nLength)
	{
		dev_YY[idx].m_nLength=dev_A->m_nLength;
	}
	for(i=0;i<dev_YY[idx].m_nLength;i++)
	{
		sum=dev_A->m_ulValue[i];
		sum=sum+dev_YY[idx].m_ulValue[i]+carry;
		dev_YY[idx].m_ulValue[i]=(unsigned long)sum;
		carry=(unsigned int)(sum>>32);
	}
	dev_YY[idx].m_ulValue[dev_YY[idx].m_nLength]=carry;
	dev_YY[idx].m_nLength+=carry;	
}


void Add_Big_Big_para(CBigInt *Y, CBigInt N, CBigInt A, int parasize)
{
    CBigInt *h_N, *h_A, *h_YY, *dev_N,*dev_A,*dev_YY;
    
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_YY,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(CBigInt));
    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_YY,parasize*(sizeof(CBigInt))));
    assignn_Big_to_Big_para(h_YY,N,32);
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_YY, h_YY, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);
    // launch the kernel
    add_big_big_thread<<<1,parasize>>>(dev_YY,dev_N,dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_YY, dev_YY, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);
    for(int i=0;i<parasize;i++){
        printf("h_YY[%d] = %s\n", i,Put(h_YY[i],HEX));
    }
    

    Mov_Big_Big(Y,h_YY[0]);
    //memcpy(Y,h_YY,parasize*(sizeof(CBigInt)));
    // free the device yo
    hipFree(h_N);
    hipFree(h_A);
    hipFree(h_YY);
    hipFree(dev_N);
    hipFree(dev_A);
    hipFree(dev_YY);
}


__global__ void add_big_long_thread(CBigInt *dev_YY, CBigInt *dev_N,unsigned long *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
    //直接在dev_YY[idx]上操作
	unsigned long sum;
	sum=dev_YY[idx].m_ulValue[0];
	sum+=(unsigned long)*dev_A;     
	dev_YY[idx].m_ulValue[0]=sum;
	if(sum>(unsigned long)0xffffffff)
	{
		unsigned int i=1;
		while(dev_YY[idx].m_ulValue[i]==0xffffffff)
		{
			dev_YY[idx].m_ulValue[i]=0;
			i++;
		}
		dev_YY[idx].m_ulValue[i]++;
		if(dev_N->m_nLength==i)
			dev_N->m_nLength++;
	}
	
}


void Add_Big_Long_para(CBigInt *Y, CBigInt N, unsigned long A, int parasize)
{
	CBigInt *h_N, *h_YY, *dev_N,*dev_YY;
    unsigned long *h_A, *dev_A;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(unsigned long),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_YY,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(unsigned long));
    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(unsigned long)));
    CHECK(hipMalloc((void **)&dev_YY,parasize*(sizeof(CBigInt))));
    //printf("ok4\n");
    assignn_Big_to_Big_para(h_YY,N,32);
    //printf("ok5\n");
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(unsigned long), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_YY, h_YY, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);
    for(int i=0;i<parasize;i++){
        printf("h_YY[%d] = %s\n", i,Put(h_YY[i],HEX));
    }
    printf("ok6\n");
    // launch the kernel
    add_big_long_thread<<<1,parasize>>>(dev_YY,dev_N,dev_A);
    // copy the result back to the CPU mem
    hipMemcpy(h_YY, dev_YY, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);
    for(int i=0;i<parasize;i++){
        printf("h_YY[%d] = %s\n", i,Put(h_YY[i],HEX));
    }
    Mov_Big_Big(Y,h_YY[0]);
    //memcpy(Y,h_YY,parasize*(sizeof(CBigInt)));
    // free the device yo
    hipFree(h_N);
    hipFree(h_A);
    hipFree(h_YY);
    hipFree(dev_N);
    hipFree(dev_A);
    hipFree(dev_YY);
    
}


/****************************************************************************************
大数相减
调用形式：Sub_Big_Big_para(Y,N,A,num) Sub_Big_Long_para(Y,N,A,num)
返回值：Y=N-A
若想返回parasize的数据量请启用 free the device yo 之前的memcpy 并注释替代函数
****************************************************************************************/
__global__ void sub_big_big_thread(CBigInt *dev_YY, CBigInt *dev_N,CBigInt *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index

	unsigned int carry=0;
	unsigned long long num;
	unsigned int i;

	if(dev_cmp(&(dev_YY[idx]),dev_A)<=0)
		dev_mov_big_long(&dev_YY[idx],0);
    else
	{
		for(i=0;i<dev_N->m_nLength;i++)
		{
			if((dev_N->m_ulValue[i]>dev_YY[idx].m_ulValue[i])||((dev_N->m_ulValue[i]==dev_YY[idx].m_ulValue[i])&&(carry==0)))
			{
				dev_YY[idx].m_ulValue[i]=dev_N->m_ulValue[i]-carry-dev_A->m_ulValue[i];
				carry=0;
			}
			else
			{
				num=0x100000000+dev_YY[idx].m_ulValue[i];
				dev_YY[idx].m_ulValue[i]=(unsigned long)(num-carry-dev_YY[idx].m_ulValue[i]);
				carry=1;
			}     
		}
		while(dev_YY[idx].m_ulValue[dev_YY[idx].m_nLength-1]==0)
			dev_YY[idx].m_nLength--;
			
	}	
}

void Sub_Big_Big_para(CBigInt *Y, CBigInt N, CBigInt A, int parasize)
{
    CBigInt *h_N, *h_A, *h_YY, *dev_N,*dev_A,*dev_YY;
    
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_YY,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    //printf("ok2\n");
    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(CBigInt));
    //printf("h_N = %s\n",Put(*h_N,HEX));
    //printf("h_A = %s\n",Put(*h_A,HEX));
    //printf("ok3\n");
    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_YY,parasize*(sizeof(CBigInt))));
    //printf("ok4\n");
    assignn_Big_to_Big_para(h_YY,N,32);
    //printf("ok5\n");
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_YY, h_YY, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);
    printf("ok6\n");
    // launch the kernel
    sub_big_big_thread<<<1,parasize>>>(dev_YY,dev_N,dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_YY, dev_YY, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    for(int i=0;i<parasize;i++){
        printf("h_YY[%d] = %s\n", i,Put(h_YY[i],HEX));
    }
    
    Mov_Big_Big(Y,h_YY[0]);
    //memcpy(Y,h_YY,parasize*(sizeof(CBigInt)));
    // free the device yo
    hipFree(h_N);
    hipFree(h_A);
    hipFree(h_YY);
    hipFree(dev_N);
    hipFree(dev_A);
    hipFree(dev_YY);
}


__global__ void sub_big_long_thread(CBigInt *dev_YY, CBigInt *dev_N,unsigned long *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
	unsigned long long num;
	int i=1;
	num=0x100000000+dev_YY[idx].m_ulValue[0];

	//Mov_Big_Big(&X,N);   因为N在之前就赋值给每条线程的dev_YY[idx]了，这里就不需要重复赋值了
	if(dev_YY[idx].m_ulValue[0]>=(*dev_A))
	{
		dev_YY[idx].m_ulValue[0]-=(*dev_A);
		
	}
	else if(dev_YY[idx].m_nLength==1)
	{
		//Mov_Big_Long(&dev_YY[idx],0);
        dev_mov_big_long(&dev_YY[idx],0);
		
	}
	else
	{

		dev_YY[idx].m_ulValue[0]=(unsigned long)(num-(*dev_A));    

		while(dev_YY[idx].m_ulValue[i]==0)
		{
			dev_YY[idx].m_ulValue[i]=0xffffffff;
			i++;
		}
		dev_YY[idx].m_ulValue[i]--;
		if(dev_YY[idx].m_ulValue[i]==0)
			dev_YY[idx].m_nLength--;
		
	}
}

void Sub_Big_Long_para(CBigInt *Y, CBigInt N, unsigned long A, int parasize)
{
    CBigInt *h_N, *h_YY, *dev_N,*dev_YY;
    unsigned long *h_A,*dev_A;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(unsigned long),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_YY,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    //printf("ok2\n");
    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(unsigned long));
    //printf("h_N = %s\n",Put(*h_N,HEX));
    //printf("h_A = %s\n",Put(*h_A,HEX));
    //printf("ok3\n");
    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(unsigned long)));
    CHECK(hipMalloc((void **)&dev_YY,parasize*(sizeof(CBigInt))));
    //printf("ok4\n");
    assignn_Big_to_Big_para(h_YY,N,32);
    //printf("ok5\n");
    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(unsigned long), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_YY, h_YY, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);
    //printf("ok6\n");
    // launch the kernel
    sub_big_long_thread<<<1,parasize>>>(dev_YY,dev_N,dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_YY, dev_YY, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    for(int i=0;i<parasize;i++){
        printf("h_YY[%d] = %s\n", i,Put(h_YY[i],HEX));
    }
    
    Mov_Big_Big(Y,h_YY[0]);
    //memcpy(Y,h_YY,parasize*(sizeof(CBigInt)));
    // free the device yo
    hipFree(h_N);
    hipFree(h_A);
    hipFree(h_YY);
    hipFree(dev_N);
    hipFree(dev_A);
    hipFree(dev_YY);	
    
    
    
}

/****************************************************************************************
大数相乘
调用形式：Mul_Big_Big(Y,N,A)
返回值：X=N*A
	A	a 0
	N	c d
		0					d*0
		1			c*0 
					d*a
		2	c*a
只有CBigInt *Y 看成是parasize个输入而CBigInt N, unsigned long A看成是一个只是并行parasize次
****************************************************************************************/
__global__ void mul_big_long_thread(CBigInt *dev_YY, CBigInt *dev_N,unsigned long *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
	unsigned long long mul;
	unsigned long carry=0;
	unsigned int i;
	//CBigIntInit(&X);
	//Mov_Big_Big(&X,N);;
	for(i=0;i<dev_N->m_nLength;i++)
	{
		mul=(unsigned long long)(dev_N->m_ulValue[i]);
		mul=mul*(*dev_A)+carry;
		dev_YY[idx].m_ulValue[i]=(unsigned int)mul;
		carry=(unsigned int)(mul>>32);
	}
	if(carry)
	{
		dev_YY[idx].m_nLength++;
		dev_YY[idx].m_ulValue[dev_YY[idx].m_nLength-1]=carry;
	}
	i = dev_YY[idx].m_nLength-1;
	while(dev_YY[idx].m_ulValue[i]==0 && i>0)
	{
		dev_YY[idx].m_nLength--;
		i--;
	}
	//Mov_Big_Big(Y,X);

}
void Mul_Big_Long_para(CBigInt *YY, CBigInt N, unsigned long A, int parasize)
{
    CBigInt *h_N, *h_YY, *dev_N,*dev_YY;
    unsigned long *h_A,*dev_A;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(unsigned long),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_YY,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
 
    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(unsigned long));
    //printf("h_N = %s\n",Put(*h_N,HEX));
    //printf("h_A = %s\n",Put(*h_A,HEX));
  
    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(unsigned long)));
    CHECK(hipMalloc((void **)&dev_YY,parasize*(sizeof(CBigInt))));
 
    assignn_Big_to_Big_para(h_YY,N,32);

    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(unsigned long), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_YY, h_YY, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    mul_big_long_thread<<<1,parasize>>>(dev_YY,dev_N,dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_YY, dev_YY, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    //Mov_Big_Big(Y,h_YY[0]);
    memcpy(YY,h_YY,parasize*(sizeof(CBigInt)));

    
    hipFree(h_N);
    hipFree(h_A);
    hipFree(h_YY);
    hipFree(dev_N);
    hipFree(dev_A);
    hipFree(dev_YY);	
   
	
}

__global__ void mul_big_big_thread(CBigInt *dev_YY, CBigInt *dev_N,CBigInt *dev_A){
    int idx = threadIdx.x; // this is how you get the thread index
	CBigInt X;
	unsigned long long sum,mul=0,carry=0;
	unsigned int i,j;
	if(dev_A->m_nLength==1)
		dev_mov_big_long(&X,dev_A->m_ulValue[0]);
	else
	{
		dev_cbigintinit(&X);
		X.m_nLength=dev_N->m_nLength+dev_A->m_nLength-1;
		for(i=0;i<X.m_nLength;i++)
		{
			sum=carry;
			carry=0;
			for(j=0;j<dev_A->m_nLength;j++)
			{
				if(((i-j)>=0)&&((i-j)<dev_N->m_nLength))
				{
					mul=dev_N->m_ulValue[i-j];
					mul*=dev_A->m_ulValue[j];
					carry+=mul>>32;
					mul=mul&0xffffffff;
					sum+=mul;
				}
			}				
			carry+=sum>>32;
			X.m_ulValue[i]=(unsigned long)sum;
		}
		if(carry)
		{
			X.m_nLength++;
			X.m_ulValue[X.m_nLength-1]=(unsigned long)carry;
		}
		i = X.m_nLength-1;
		while(X.m_ulValue[i]==0 && i>0)
		{
			X.m_nLength--;
			i--;
		}
		dev_mov_big_big(&dev_YY[idx],X);
	}

}

void Mul_Big_Big_para(CBigInt *YY, CBigInt N, CBigInt A, int parasize)
{
    CBigInt *h_N, *h_A, *h_YY, *dev_N, *dev_A, *dev_YY;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_YY,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
 
    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(CBigInt));
    //printf("h_N = %s\n",Put(*h_N,HEX));
    //printf("h_A = %s\n",Put(*h_A,HEX));
  
    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_YY,parasize*(sizeof(CBigInt))));
 
    //assignn_Big_to_Big_para(h_YY,N,32);

    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    //hipMemcpy(dev_YY, h_YY, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    mul_big_big_thread<<<1,parasize>>>(dev_YY,dev_N,dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_YY, dev_YY, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    //Mov_Big_Big(Y,h_YY[0]);
    memcpy(YY,h_YY,parasize*(sizeof(CBigInt)));
    /*
    for(int i=0;i<parasize;i++){
        printf("thread work out YY[%d] = %s\n",i,Put(YY[i],HEX));
    }
    */
    
    hipFree(h_N);
    hipFree(h_A);
    hipFree(h_YY);
    hipFree(dev_N);
    hipFree(dev_A);
    hipFree(dev_YY);	

}


/****************************************************************************************
大数相除
调用形式：Div_Big_Big_para(P,N,A)
返回值：P=N/A
****************************************************************************************/
__global__ void div_big_long_thread(CBigInt *dev_YY, CBigInt *dev_N,unsigned long *dev_A){
    int idx = threadIdx.x; 
    //CBigInt X;
	unsigned long long div,mul;
	unsigned long carry=0;
	int i;
	//CBigIntInit(X);
	//Mov_Big_Big(&X,N);;
	if(dev_YY[idx].m_nLength==1)
	{
		dev_YY[idx].m_ulValue[0]=dev_YY[idx].m_ulValue[0]/(*dev_A);
		//Mov_Big_Big(Y,dev_YY[idx]);
	}
	else
	{
		for(i=dev_YY[idx].m_nLength-1;i>=0;i--)
		{
			div=carry;
			div=(div<<32)+dev_YY[idx].m_ulValue[i];
			dev_YY[idx].m_ulValue[i]=(unsigned long)(div/(*dev_A));
			mul=(div/(*dev_A))*(*dev_A);
			carry=(unsigned long)(div-mul);
		}
		if(dev_YY[idx].m_ulValue[dev_YY[idx].m_nLength-1]==0)
			dev_YY[idx].m_nLength--;
		//Mov_Big_Big(Y,X);
	}

}

void Div_Big_Long_para(CBigInt *YY, CBigInt N, unsigned long A, int parasize)
{
	CBigInt *h_N, *h_YY, *dev_N,*dev_YY;
    unsigned long *h_A,*dev_A;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(unsigned long),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_YY,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
 
    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(unsigned long));
    //printf("h_N = %s\n",Put(*h_N,HEX));
    //printf("h_A = %s\n",Put(*h_A,HEX));
  
    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(unsigned long)));
    CHECK(hipMalloc((void **)&dev_YY,parasize*(sizeof(CBigInt))));
 
    assignn_Big_to_Big_para(h_YY,N,32);

    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(unsigned long), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_YY, h_YY, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    div_big_long_thread<<<1,parasize>>>(dev_YY,dev_N,dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_YY, dev_YY, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    //Mov_Big_Big(Y,h_YY[0]);
    memcpy(YY,h_YY,parasize*(sizeof(CBigInt)));

    
    hipFree(h_N);
    hipFree(h_A);
    hipFree(h_YY);
    hipFree(dev_N);
    hipFree(dev_A);
    hipFree(dev_YY);
    

}

__global__ void div_big_big_thread(CBigInt *dev_YY, CBigInt *dev_N,CBigInt *dev_A){
    int idx = threadIdx.x;        
    CBigInt X,Y,Z,T;
    unsigned int i,len;
    unsigned long long num,div;
    dev_cbigintinit(&X);
    dev_cbigintinit(&Y);
    dev_cbigintinit(&Z);
    dev_cbigintinit(&T); //dev_cbigintinit
    dev_mov_big_big(&Y,*dev_N); //dev_mov_big_big
    while(dev_cmp(&Y,dev_A)>=0)
    {        
        div=Y.m_ulValue[Y.m_nLength-1];
        num=dev_A->m_ulValue[dev_A->m_nLength-1];
        len=Y.m_nLength-dev_A->m_nLength;
        if((div==num)&&(len==0))
        {
            //Mov_Big_Big(&X,Add_Big_Long(X,1));
            dev_add_big_long(&X,X,1); //dev_add_big_long
            break;
        }
        if((div<=num)&&len)
        {
            len--;
            div=(div<<32)+Y.m_ulValue[Y.m_nLength-2];
        }
        div=div/(num+1);
        dev_mov_big_long(&Z,div); //dev_mov_big_long
        if(len)
        {
            Z.m_nLength+=len;
            for(i=Z.m_nLength-1;i>=len;i--)
                Z.m_ulValue[i]=Z.m_ulValue[i-len];
            for(i=0;i<len;i++)
                Z.m_ulValue[i]=0;
        }
        //Mov_Big_Big(&X,Add_Big_Big(X,Z));
        dev_add_big_big(&X,X,Z); //dev_add_big_big
        //Mov_Big_Big(&Y,Sub_Big_Big(Y,Mul_Big_Big(A,Z)));
        dev_mul_big_big(&T,*dev_A,Z);
        dev_sub_big_big(&Y,Y,T);
    }
    dev_mov_big_big(&(dev_YY[idx]),X);

}


void Div_Big_Big_para(CBigInt *MM,CBigInt N, CBigInt A,int parasize)
{
	if(A.m_nLength==1)
		Div_Big_Long_para(MM,N,A.m_ulValue[0],parasize);
	else
	{
        //emmm 直接把下面的函数放入到kernal岂不是很蠢吗 不管了 先这样做着
        CBigInt *h_N, *h_YY, *dev_N,*dev_YY;
        CBigInt *h_A,*dev_A;
        // host alloc and cuda malloc in one time
        CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
        CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));
        CHECK(hipHostAlloc((void**) &h_YY,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    
        memcpy(h_N,&N,sizeof(CBigInt));
        memcpy(h_A,&A,sizeof(CBigInt));
        //printf("h_N = %s\n",Put(*h_N,HEX));
        //printf("h_A = %s\n",Put(*h_A,HEX));
    
        CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
        CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
        CHECK(hipMalloc((void **)&dev_YY,parasize*(sizeof(CBigInt))));
    
        assignn_Big_to_Big_para(h_YY,N,32);

        // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
        hipMemcpy(dev_A, h_A, sizeof(CBigInt), hipMemcpyHostToDevice);
        hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
        hipMemcpy(dev_YY, h_YY, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

        // launch the kernel
        div_big_big_thread<<<1,parasize>>>(dev_YY,dev_N,dev_A);

        // copy the result back to the CPU mem
        hipMemcpy(h_YY, dev_YY, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

        //Mov_Big_Big(Y,h_YY[0]);
        memcpy(MM,h_YY,parasize*(sizeof(CBigInt)));

        
        hipFree(h_N);
        hipFree(h_A);
        hipFree(h_YY);
        hipFree(dev_N);
        hipFree(dev_A);
        hipFree(dev_YY);

        
	}
}


/****************************************************************************************
大数求模
调用形式：Mod_Big_Big_para(Z,N,A,num)
返回值：Z=N%A
****************************************************************************************/
__global__ void mod_big_big_thread(CBigInt *dev_YY, CBigInt *dev_N,CBigInt *dev_A){
    int idx = threadIdx.x;    
    CBigInt X,Y;
	unsigned long long div,num;
	//unsigned long carry=0;
	unsigned int i,len;
	dev_cbigintinit(&X);
	dev_cbigintinit(&Y);
	dev_mov_big_big(&X,*dev_N);
	while(dev_cmp(&X,dev_A)>=0)
	{
		div=X.m_ulValue[X.m_nLength-1];
		num=dev_A->m_ulValue[dev_A->m_nLength-1];
		len=X.m_nLength-dev_A->m_nLength;
		if((div==num)&&(len==0))
		{
			//Mov_Big_Big(&X,Sub_Big_Big(X,A));
			dev_sub_big_big(&X,X,*dev_A);
			break;
		}
		if((div<=num)&&len)
		{
			len--;div=(div<<32)+X.m_ulValue[X.m_nLength-2];
		}
		div=div/(num+1);
		dev_mov_big_long(&Y,div);		
		//Mov_Big_Big(&Y,Mul_Big_Big(A,Y));
		dev_mul_big_big(&Y,*dev_A,Y);
		if(len)
		{
			Y.m_nLength+=len;
			for(i=Y.m_nLength-1;i>=len;i--)
				Y.m_ulValue[i]=Y.m_ulValue[i-len];
			for(i=0;i<len;i++)
				Y.m_ulValue[i]=0;
		}
		//Mov_Big_Big(&X,Sub_Big_Big(X,Y));
		dev_sub_big_big(&X,X,Y);
	}
	dev_mov_big_big(&(dev_YY[idx]),X);

}

void Mod_Big_Big_para(CBigInt *ZZ,CBigInt N, CBigInt A, int parasize)
{
    CBigInt *h_N, *h_YY, *dev_N,*dev_YY;
    CBigInt *h_A,*dev_A;
    // host alloc and cuda malloc in one time
    CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_YY,parasize*(sizeof(CBigInt)),hipHostMallocDefault));

    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(CBigInt));
    //printf("h_N = %s\n",Put(*h_N,HEX));
    //printf("h_A = %s\n",Put(*h_A,HEX));

    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_YY,parasize*(sizeof(CBigInt))));

    //assignn_Big_to_Big_para(h_YY,N,32);

    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    //hipMemcpy(dev_YY, h_YY, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    mod_big_big_thread<<<1,parasize>>>(dev_YY,dev_N,dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_YY, dev_YY, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    //Mov_Big_Big(Y,h_YY[0]);
    memcpy(ZZ,h_YY,parasize*(sizeof(CBigInt)));

    
    hipFree(h_N);
    hipFree(h_A);
    hipFree(h_YY);
    hipFree(dev_N);
    hipFree(dev_A);
    hipFree(dev_YY);

}
__global__ void mod_big_big_thread(unsigned long *ZZ, CBigInt *N, unsigned long *A){
    int idx = threadIdx.x;   
	unsigned long long div;
	unsigned long carry=0;
	int i;
	if(N->m_nLength==1){
        ZZ[idx] = N->m_ulValue[0]%(*A);
        return;
    }
		

	for(i=N->m_nLength-1;i>=0;i--)
	{
		div=N->m_ulValue[i];
		div+=carry*0x100000000;
		carry=(unsigned long)(div%(*A));
	}
    ZZ[idx] = carry;
	return;
}

void Mod_Big_Long_para(unsigned long *ZZ, CBigInt N, unsigned long A,int parasize)
{
	CBigInt *h_N, *dev_N;
    unsigned long *h_A,*dev_A,*h_ZZ,*dev_ZZ;
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(unsigned long),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_ZZ,parasize*(sizeof(unsigned long)),hipHostMallocDefault));
 
    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(unsigned long));
    //printf("h_N = %s\n",Put(*h_N,HEX));
    //printf("h_A = %s\n",Put(*h_A,HEX));
  
    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(unsigned long)));
    CHECK(hipMalloc((void **)&dev_ZZ,parasize*(sizeof(unsigned long))));

    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(unsigned long), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    //hipMemcpy(dev_ZZ, h_ZZ, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    mod_big_big_thread<<<1,parasize>>>(dev_ZZ,dev_N,dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_ZZ, dev_ZZ, parasize*(sizeof(unsigned long)), hipMemcpyDeviceToHost);

    //Mov_Big_Big(Y,h_YY[0]);
    memcpy(ZZ,h_ZZ,parasize*(sizeof(unsigned long)));

    
    hipFree(h_N);
    hipFree(h_A);
    hipFree(h_ZZ);
    hipFree(dev_N);
    hipFree(dev_A);
    hipFree(dev_ZZ);
}


/****************************************************************************************
从字符串按10进制或16进制格式输入到 N个 大数
调用格式：Get(N,str,sys)
返回值：N被赋值为相应大数
sys暂时只能为10或16
****************************************************************************************/
__global__ void get_para_thread(CBigInt *dev_NN, char* dev_str, unsigned int dev_system, int str_len){
    int idx = threadIdx.x;   
    int i;
	int k;
	dev_cbigintinit(&(dev_NN[idx]));

	
	dev_mov_big_long(&(dev_NN[idx]),0);

	for(i = 0; i < str_len; i++)
	{
		dev_mul_big_long(&(dev_NN[idx]),dev_NN[idx],(unsigned long)dev_system);
		if((dev_str[i]>='0')&&(dev_str[i]<='9'))
			k=dev_str[i]-48;
		else 
			if((dev_str[i]>='A')&&(dev_str[i]<='F'))
				k=dev_str[i]-55;
		else 
			if((dev_str[i]>='a')&&(dev_str[i]<='f'))
				k=dev_str[i]-87;
		else k=0;

		dev_add_big_long(&(dev_NN[idx]),dev_NN[idx],k);

	}


}
void Get_para(CBigInt *NN, char* str, unsigned int system, int parasize)
{
    
    CBigInt *h_NN, *dev_NN;
    char *h_str,*dev_str;
    int h_len = strlen(str);
    
    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_NN,parasize*sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_str,1024,hipHostMallocDefault));

    strcpy(h_str,str);

  
    CHECK(hipMalloc((void **)&dev_NN,parasize*sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_str,1024));

    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_str, h_str, 1024, hipMemcpyHostToDevice);


    // launch the kernel
    get_para_thread<<<1,parasize>>>(dev_NN,dev_str,system,h_len);

    // copy the result back to the CPU mem
    hipMemcpy(h_NN, dev_NN, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    memcpy(NN,h_NN,parasize*(sizeof(CBigInt)));
    
    hipFree(h_NN);
    hipFree(h_str);
    hipFree(dev_NN);
    hipFree(dev_str);
	
}

/****************************************************************************************
将大数按10进制或16进制格式输出为字符串
调用格式：Put_para(N,str,sys,num)
返回值：无，直接GPU打印多个大数为16或者10进制字符串 
sys暂时只能为10或16
****************************************************************************************/
__global__ void put_para_thread(CBigInt *dev_NN, unsigned int dev_system){
    int idx = threadIdx.x;
    printf("dev_NN[%d] CBigInt = %s\n",idx,dev_put(&(dev_NN[idx]),dev_system));
}

void Put_para(CBigInt *NN, unsigned int system,int parasize)
{
    CBigInt *dev_NN;

    // host alloc and cuda malloc in one time
    CHECK(hipMalloc((void **)&dev_NN,parasize*sizeof(CBigInt)));


    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_NN, NN, parasize*sizeof(CBigInt), hipMemcpyHostToDevice);

    // launch the kernel
    put_para_thread<<<1,parasize>>>(dev_NN,system);
    // copy the result back to the CPU mem
    hipFree(dev_NN);
	
}


/****************************************************************************************
求不定方程ax-by=1的最小整数解
调用方式：Inv_para(N,A,num)
返回值：Z,满足：NZ mod A=1
****************************************************************************************/
__global__ void inv_para_thread(CBigInt *dev_NN, CBigInt *dev_N, CBigInt *dev_A){
    int idx = threadIdx.x;
	CBigInt M,E,X,Y,I,J;
	int x,y;	
	dev_cbigintinit(&M);
	dev_cbigintinit(&E);
	dev_cbigintinit(&X);
	dev_cbigintinit(&Y);
	dev_cbigintinit(&I);
	dev_cbigintinit(&J);
	if(dev_cmp(dev_N,dev_A)>=0)//dev_cmp
		dev_mod_big_big(dev_N,*dev_N,*dev_A);//dev_mod_big_big
	dev_mov_big_big(&M,*dev_A);
	dev_mov_big_big(&E,*dev_N);
	dev_mov_big_long(&X,0); //dev_mov_big_long
	dev_mov_big_long(&Y,1);
	x=y=1;
    //printf("Para Test Cbitint  dev_A\n");
    //printf("Para Test Cbitint  dev_A= %s\n",dev_put(dev_A,HEX)); //Put(YY[12],HEX)
	while((E.m_nLength!=1)||(E.m_ulValue[0]!=0))
	{
		//Mov_Big_Big(&I,Div_Big_Big(M,E));
		dev_div_big_big(&I,M,E); //dev_div_big_big
		//Mov_Big_Big(&J,Mod_Big_Big(M,E));
		dev_mod_big_big(&J,M,E);//dev_mod_big_big
		dev_mov_big_big(&M,E);
		dev_mov_big_big(&E,J);
		dev_mov_big_big(&J,Y);
		//Mov_Big_Big(&Y,Mul_Big_Big(Y,I));
		dev_mul_big_big(&Y,Y,I); //dev_mul_big_big
		if(x==y)
		{
			if(dev_cmp(&X,&Y)>=0)
				dev_sub_big_big(&Y,X,Y); // Mov_Big_Big(&Y,Sub_Big_Big(X,Y));
			else
			{
				//Mov_Big_Big(&Y,Sub_Big_Big(Y,X));
				dev_sub_big_big(&Y,Y,X);
				y=0;
			}
		}
		else
		{
			//Mov_Big_Big(&Y,Add_Big_Big(X,Y));
			dev_add_big_big(&Y,X,Y);
			x=1-x;
			y=1-y;
		}
		dev_mov_big_big(&X,J);
	}
	if(x==0)
		dev_sub_big_big(&X,*dev_A,X);  // Mov_Big_Big(&X,Sub_Big_Big(A,X));
	if(dev_cmp(&X,dev_A)>= 0)
	    dev_mod_big_big(&X,X,*dev_A);
	dev_mov_big_big(&(dev_NN[idx]),X); //dev_mov_big_big
}

void Inv_para(CBigInt *ZZ, CBigInt N, CBigInt A, int parasize)
{
    CBigInt *h_N, *dev_N,*dev_ZZ;
    CBigInt *h_A,*dev_A;
    // host alloc and cuda malloc in one time
    CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));
    //CHECK(hipHostAlloc((void**) &h_ZZ,parasize*(sizeof(CBigInt)),hipHostMallocDefault));

    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(CBigInt));
    //printf("h_N = %s\n",Put(*h_N,HEX));
    //printf("h_A = %s\n",Put(*h_A,HEX));

    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_ZZ,parasize*(sizeof(CBigInt))));

    //assignn_Big_to_Big_para(h_YY,N,32);

    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    //hipMemcpy(dev_ZZ, h_ZZ, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    inv_para_thread<<<1,parasize>>>(dev_ZZ,dev_N,dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(ZZ, dev_ZZ, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    //Mov_Big_Big(Y,h_YY[0]);
    //memcpy(ZZ,h_YY,parasize*(sizeof(CBigInt)));

    
    hipFree(h_N);
    hipFree(h_A);
    //hipFree(h_YY);
    hipFree(dev_N);
    hipFree(dev_A);
    hipFree(dev_ZZ);


}

/****************************************************************************************
求乘方的模
调用方式：Exp_para(N,A,B,num)
返回值：Z=N^A MOD B
****************************************************************************************/
__global__ void exp_para_thread(CBigInt *dev_NN, CBigInt *dev_N, CBigInt *dev_A,  CBigInt *dev_B){
    int idx = threadIdx.x;
    CBigInt X,Y,M;
	int i,j,k;
	unsigned int n;
	unsigned long num;
	k=dev_A->m_nLength*32-32;
	num=dev_A->m_ulValue[dev_A->m_nLength-1];
	while(num)
	{
		num=num>>1;
		k++;
	}
	dev_cbigintinit(&X);
	dev_cbigintinit(&Y);
	dev_cbigintinit(&M);
	dev_mov_big_big(&X,*dev_N);
	for(i=k-2;i>=0;i--)
	{
		//Mov_Big_Big(&Y,Mul_Big_Long(X,X.m_ulValue[X.m_nLength-1]));
		dev_mul_big_long(&Y,X,X.m_ulValue[X.m_nLength-1]); //dev_mul_big_long
		//Mov_Big_Big(&Y,Mod_Big_Big(Y,B));
		dev_mod_big_big(&Y,Y,*dev_B); //dev_mod_big_big
		for(n=1;n<X.m_nLength;n++)
		{          
			for(j=Y.m_nLength;j>0;j--)
				Y.m_ulValue[j]=Y.m_ulValue[j-1];
			Y.m_ulValue[0]=0;
			Y.m_nLength++;
			//Mov_Big_Big(&Y,Add_Big_Big(Y,Mul_Big_Long(X,X.m_ulValue[X.m_nLength-n-1])));
			dev_mul_big_long(&Y,X,X.m_ulValue[X.m_nLength-n-1]);
			//Mov_Big_Big(&Y,Mod_Big_Big(Y,B));
			dev_mod_big_big(&Y,Y,*dev_B);
		}
		dev_mov_big_big(&X,Y);
		if((dev_A->m_ulValue[i>>5]>>(i&31))&1)
		{
			//Mov_Big_Big(&Y,Mul_Big_Long(N,X.m_ulValue[X.m_nLength-1]));
			dev_mul_big_long(&Y,*dev_N,X.m_ulValue[X.m_nLength-1]);
			//Mov_Big_Big(&Y,Mod_Big_Big(Y,B));
			dev_mod_big_big(&Y,Y,*dev_B);
			for(n=1;n<X.m_nLength;n++)
			{          
				for(j=Y.m_nLength;j>0;j--)
					Y.m_ulValue[j]=Y.m_ulValue[j-1];
				Y.m_ulValue[0]=0;
				Y.m_nLength++;
				//Mov_Big_Big(&Y,Add_Big_Big(Y,Mul_Big_Long(N,X.m_ulValue[X.m_nLength-n-1])));
				dev_mul_big_long(&M,*dev_N,X.m_ulValue[X.m_nLength-n-1]);
				dev_add_big_big(&Y,Y,M);
				//Mov_Big_Big(&Y,Mod_Big_Big(Y,B));
				dev_mod_big_big(&Y,Y,*dev_B);
			}
			dev_mov_big_big(&X,Y);
		}
	}
	dev_mov_big_big(&(dev_NN[idx]),X);

}

void Exp_para(CBigInt *ZZ, CBigInt N, CBigInt A, CBigInt B, int parasize)
{

    CBigInt *h_N, *dev_N,*dev_ZZ,*h_A,*dev_A,*h_B,*dev_B;;
    // host alloc and cuda malloc in one time
    CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));
    CHECK(hipHostAlloc((void**) &h_B,sizeof(CBigInt),hipHostMallocDefault));
    //CHECK(hipHostAlloc((void**) &h_ZZ,parasize*(sizeof(CBigInt)),hipHostMallocDefault));

    memcpy(h_N,&N,sizeof(CBigInt));
    memcpy(h_A,&A,sizeof(CBigInt));
    memcpy(h_B,&B,sizeof(CBigInt));
    //printf("h_N = %s\n",Put(*h_N,HEX));
    //printf("h_A = %s\n",Put(*h_A,HEX));

    CHECK(hipMalloc((void **)&dev_N,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_A,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_B,sizeof(CBigInt)));
    CHECK(hipMalloc((void **)&dev_ZZ,parasize*(sizeof(CBigInt))));

    //assignn_Big_to_Big_para(h_YY,N,32);

    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_A, h_A, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_N, h_N, sizeof(CBigInt), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, h_B, sizeof(CBigInt), hipMemcpyHostToDevice);
    //hipMemcpy(dev_ZZ, h_ZZ, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    exp_para_thread<<<1,parasize>>>(dev_ZZ,dev_N,dev_A,dev_B);

    // copy the result back to the CPU mem
    hipMemcpy(ZZ, dev_ZZ, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    //Mov_Big_Big(Y,h_YY[0]);
    //memcpy(ZZ,h_YY,parasize*(sizeof(CBigInt)));

    
    hipFree(h_N);
    hipFree(h_A);
    hipFree(h_B);
    //hipFree(h_YY);
    hipFree(dev_N);
    hipFree(dev_B);
    hipFree(dev_A);
    hipFree(dev_ZZ);

}
