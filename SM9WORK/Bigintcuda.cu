#include "hip/hip_runtime.h"
#include "Bigintcuda.cuh"



//构造大数对象并初始化为零 + 其对应的kernal函数

__global__ void cbiginiinit_thread(CBigInt *d_out){
    int idx = threadIdx.x; // this is how you get the thread index
	d_out[idx].m_nLength=1;
	for(int i=0;i<BI_MAXLEN;i++)
		d_out[idx].m_ulValue[i]=0;
}

void CBigIntInit_para(CBigInt *A,int parasize)
{
    CBigInt *h_A, *dev_A;


    // host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_A,parasize*(sizeof(CBigInt)),hipHostMallocDefault));
    //数据从*A 拷贝到 h_A

    // copy from host memory to pinned memory
	CHECK(hipMalloc((void **)&dev_A,parasize*(sizeof(CBigInt))));

    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    //hipMemcpy(dev_A, h_A, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);

    // launch the kernel
    cbiginiinit_thread<<<1,parasize>>>(dev_A);

    // copy the result back to the CPU mem
    hipMemcpy(h_A, dev_A, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    // print the result
    
    for (int i = 0; i < parasize; i++){
        printf("para %d = %s\n",i,Put(h_A[i],HEX));
    }

    Mov_Big_Big(A,h_A[0]);

    // free the device yo
    hipFree(h_A);
    hipFree(dev_A);
}





