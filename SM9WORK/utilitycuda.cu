#include "hip/hip_runtime.h"
#include "Bigintcuda.cuh"

#include "Params.h"

__global__ void cintbig_substract_modn_thread(CBigInt *dev_X, CBigInt *dev_a,CBigInt *dev_b,CBigInt *dev_BNn){
    int idx = threadIdx.x;    
    
    CBigInt c,div;
	if(dev_cmp(&dev_a[idx],&dev_b[idx])>=0)
	{
		dev_sub_big_big(&c,dev_a[idx],dev_b[idx]);  //
		if(dev_cmp(&c,dev_BNn)>=0 )
	      dev_mod_big_big(&c,c,*dev_BNn);
	}
    else
	{
		//
		dev_sub_big_big(&c,dev_b[idx],dev_a[idx]);		
		dev_div_big_big(&div,c,*dev_BNn);
		dev_add_big_long(&div,div,1);
		dev_mul_big_big(&div,div,*dev_BNn);
		dev_sub_big_big(&c,div,c);
		if(dev_cmp(&c,dev_BNn)==0)
			dev_mov_big_long(&c,0);
	}

	dev_mov_big_big(&(dev_X[idx]),c);

}


void CBigInt_substract_modN_para(CBigInt *XX, CBigInt *aa, CBigInt *bb,int parasize){
	


    CBigInt *dev_XX,*dev_aa,*dev_bb,*dev_BNn;
    
    // host alloc and cuda malloc in one time
    //CHECK(hipHostAlloc((void**) &h_N,sizeof(CBigInt),hipHostMallocDefault));
    //CHECK(hipHostAlloc((void**) &h_A,sizeof(CBigInt),hipHostMallocDefault));
    //CHECK(hipHostAlloc((void**) &h_YY,parasize*(sizeof(CBigInt)),hipHostMallocDefault));

    //memcpy(h_N,&N,sizeof(CBigInt));
    //memcpy(h_A,&A,sizeof(CBigInt));
    //printf("h_N = %s\n",Put(*h_N,HEX));
    //printf("h_A = %s\n",Put(*h_A,HEX));

    CHECK(hipMalloc((void **)&dev_aa,parasize*(sizeof(CBigInt))));
    CHECK(hipMalloc((void **)&dev_bb,parasize*(sizeof(CBigInt))));
    CHECK(hipMalloc((void **)&dev_XX,parasize*(sizeof(CBigInt))));
    CHECK(hipMalloc((void **)&dev_BNn,sizeof(CBigInt)));
    

    //assignn_Big_to_Big_para(h_YY,N,32);

    // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
    hipMemcpy(dev_XX, XX, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);
    hipMemcpy(dev_aa, aa, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);
    hipMemcpy(dev_bb, bb, parasize*(sizeof(CBigInt)), hipMemcpyHostToDevice);
    hipMemcpy(dev_BNn, &BN.n, sizeof(CBigInt), hipMemcpyHostToDevice);
    // launch the kernel
    cintbig_substract_modn_thread<<<1,parasize>>>(dev_XX,dev_aa,dev_bb, dev_BNn);

    // copy the result back to the CPU mem
    hipMemcpy(XX, dev_XX, parasize*(sizeof(CBigInt)), hipMemcpyDeviceToHost);

    //Mov_Big_Big(Y,h_YY[0]);
    //memcpy(ZZ,h_YY,parasize*(sizeof(CBigInt)));

    
    //hipFree(h_N);
    //hipFree(h_A);
    //hipFree(h_YY);
    hipFree(dev_bb);
    hipFree(dev_aa);
    hipFree(dev_XX);
}